#include "hip/hip_runtime.h"
#include "Estimator.cuh"
#include "../utils/Dataset.h"
#include <iostream>
#include <math.h>

__global__ void partialMoments(float *x, float* moments, int n){
	extern __shared__ float sdata[];
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x * 2,
				 total = blockDim.x  * gridDim.x,
				 thidx = threadIdx.x;

    if(index < n)
    	sdata[thidx] = x[index] + x[index + blockDim.x];
    else
    	sdata[thidx] = (powf(x[index - total], 2) +  powf(x[index + blockDim.x - total], 2));

    __syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if(thidx < s){
			sdata[thidx] += sdata[thidx + s];
			__syncthreads();
		}
	}
	if(thidx == 0)
		moments[blockIdx.x] = sdata[thidx];
}


__global__ void composeMoments(float * moments, int block, int n){
	float mean=0, var=0; int stride = block/2;
	for (int i = 0; i < stride; ++i) {
		mean += moments[i];
		var += moments[i + stride];
	}
	moments[0] = mean/n; moments[stride] = sqrtf(var/n - powf(mean/n, 2));
}

__global__ void standardNormalization(float *X, float *moments, int stride){
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	X[index] = (X[index] - moments[0])/moments[stride];
}

__global__ void fillGramMatrix(float* in, float* out,
		float sigma, int n, int m){

	__shared__ float Ys[16][16];
	__shared__ float Xs[16][16];

	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;

	int yBegin = by * 16 * m;
	int xBegin = bx * 16 * m;

	int yEnd = yBegin + m - 1, y, x, k, o;

	float tmp, s = 0;

	for (y = yBegin, x = xBegin; y <= yEnd;	y += 16, x += 16){
		Ys[ty][tx] = in[y + ty * m + tx];
		Xs[tx][ty] = in[x + ty * m + tx];

		__syncthreads();

		for (k = 0; k<16; k++){
			tmp = Ys[ty][k] - Xs[k][tx];
			s += tmp * tmp;
		}
		__syncthreads();
	}
	o = by * 16 * n + ty * n + bx * 16 + tx;

	out[o] = exp(- s / (2 * pow(sigma, 2)));
}

__global__ void hadamardProduct(float *X, float *Y, float *XY){

}


namespace Estimator {

	unsigned int computeBlocks(int threads, int n){
		unsigned int blocks = (n + threads - 1)/threads;
		blocks = (blocks % 2 == 0) ? blocks : blocks + 1;
		return blocks;
	}

	float getSigma(int rows, int cols) {
		float penalizer = -1.0/(4 + cols);
		float scottFactor = pow(rows, penalizer);
		return sqrt(2 * cols) * scottFactor;
	}


	float computeInformationTheoryParallel(float *X, float *Y,
			int rows, int xcols, int ycols){ checkAvailableDevices();

		const unsigned int threads = 1024; size_t sfloat = sizeof(float);
		unsigned int blocks = computeBlocks(threads, rows * xcols);

		//////////////////////////// Normalizing X ////////////////////////////

		size_t sizex = sfloat * rows * xcols; float *Xdev, *momentsx;

		hipMalloc((void **) &Xdev, sizex);
		hipMalloc((void **) &momentsx, sfloat * blocks);

		hipMemcpy(Xdev, X, sizex, hipMemcpyHostToDevice);

		partialMoments<<<blocks, threads, sfloat * threads>>>(
				Xdev, momentsx, rows * xcols);

		composeMoments<<<1, 1>>>(momentsx, blocks, rows * xcols);

		standardNormalization<<<blocks, threads>>>(
				Xdev, momentsx, blocks / 2);

		//////////////////////////// Normalizing Y ////////////////////////////

		size_t sizey = sfloat * rows * ycols; float *Ydev, *momentsy;
		blocks = computeBlocks(threads, rows * ycols);

		hipMalloc((void **) &Ydev, sizey);
		hipMalloc((void **) &momentsy, sfloat * blocks);

		hipMemcpy(Ydev, Y, sizey, hipMemcpyHostToDevice);

		partialMoments<<<blocks, threads, sfloat * threads>>>(
				Ydev, momentsy, rows * ycols);

		composeMoments<<<1, 1>>>(momentsy, blocks, rows * ycols);

		standardNormalization<<<blocks, threads>>>(
				Ydev, momentsy, blocks / 2);

		//////////////////////////// Compute Gram Matrix ///////////////////

		float * gramX, * gramY; size_t sgram = sfloat * pow(rows, 2);

		hipMalloc((void **) &gramX, sgram);
		hipMalloc((void **) &gramY, sgram);

		blocks = computeBlocks(threads, pow(rows, 2));

		dim3 block(16, 16);
		dim3 grid(ceil(rows/16), ceil(rows/16));

		fillGramMatrix<<<grid, block >>>(Xdev, gramX,
				getSigma(rows, xcols), rows, xcols);

		fillGramMatrix<<<grid, block >>>(Ydev, gramY,
				getSigma(rows, ycols), rows, ycols);


		//////////////////////////// Compute Eigen Values & Entropy///////////////////





		//////////////////////////// Compute Joint Entropy ///////////////////






		//////////////////////////// Compute Mutual Information ///////////////////



	    hipDeviceSynchronize();

	    float mutualInformation;



		hipFree(Xdev); hipFree(momentsx);
		hipFree(gramX); hipFree(gramY);
		hipFree(Ydev); hipFree(momentsy);


		return mutualInformation;
	}

	void checkAvailableDevices(){
		int deviceCount; hipGetDeviceCount(&deviceCount);
		if(deviceCount == 0){
			fprintf(stderr, "Error: No devices supporting CUDA.\n");
			exit(EXIT_FAILURE);
		}
	}

}


